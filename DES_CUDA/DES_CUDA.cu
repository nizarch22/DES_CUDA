#include "hip/hip_runtime.h"
#define __HIPCC__
#include <cstdlib>
// External
#include "hip/hip_runtime.h"
#include ""
#include "DES_CUDA.cuh"
// Primary header is compatible with pre-C++11, collective algorithm headers require C++11
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;
__device__ void swapLRCuda(unsigned char* input, unsigned char* sharedCopy); // Swap left (32 bit) and right (32 bit) parts of the 64 bit input.
__device__ void substituteCuda(unsigned char* input, uint16_t* sharedX, uint16_t* sharedY, const unsigned char* d_SBoxesConst);
__device__ void leftCircularShiftCuda(unsigned char* input, unsigned char* sharedCopy, uint8_t times);
__device__ void rightCircularShiftCuda(unsigned char* input, unsigned char* sharedCopy, uint8_t times);
__device__ void permuteMatrixCuda(unsigned char* input, unsigned char* sharedCopy, const unsigned char* P, const unsigned int size);


__device__ void copy(unsigned char* debug, unsigned char* target, int num)
{
	debug[threadIdx.x + num*64 + blockIdx.x * 150*64] = target[threadIdx.x];
	__syncthreads();
}

__global__ void EncryptDESCuda(uint64_t* messages, uint64_t* keys, uint64_t* results, const unsigned char* d_matricesConst, const unsigned char* d_SBoxesConst)
{
	// Kernel iterations shared memory
	__shared__ unsigned char sharedInput[128];
	__shared__ unsigned char sharedLeft[128];
	__shared__ unsigned char sharedResult[128];
	__shared__ unsigned char sharedKey[128];
	__shared__ unsigned char sharedRoundkey[128];
	__shared__ uint64_t result; // setting alias for encryption

	// General shared array. Typically for copying input. Used in the following functions: permuteMatrixCuda, swapLRCuda, leftCircularShiftCuda, rightCircularShiftCuda
	__shared__ unsigned char sharedCopy[128];
	// Special arrays for 'subsituteCuda' function:
	__shared__ uint16_t sharedX[16];
	__shared__ uint16_t sharedY[16];

	uint64_t input;
	uint64_t shiftedKey;
	const int matricesIndices[7] = { 0, 64,120,168,216,248,312 };

	int threadSetIndex = threadIdx.x >> 6; // divide by 64=2^6
	int threadIndex = (threadIdx.x >= 64) ? (threadIdx.x - 64) : threadIdx.x;

	// Initializations
	input = messages[(blockIdx.x << 1) + threadSetIndex];
	shiftedKey = keys[(blockIdx.x << 1) + threadSetIndex];
	sharedInput[threadIdx.x] = 0;
	sharedLeft[threadIdx.x] = 0;
	sharedResult[threadIdx.x] = 0;
	sharedKey[threadIdx.x] = 0;
	sharedRoundkey[threadIdx.x] = 0;
	sharedCopy[threadIdx.x] = 0;
	if (threadIndex < 8)
	{
		sharedX[threadIdx.x] = 0;
		sharedY[threadIdx.x] = 0;
	}
	if (threadIndex == 0)
	{
		result = 0;
	}
	__syncthreads();

	// Initial operations 
	// The 64 bits of message,key (uint64_t) are converted into 64 bytes (unsigned char) so that they are easily parallelizable. 
	sharedInput[threadIdx.x] = (input >> threadIndex) & 1;
	sharedKey[threadIdx.x] = (shiftedKey >> threadIndex) & 1;
	__syncthreads();

	// Initial permutation parallelized
	permuteMatrixCuda(sharedInput, sharedCopy, &d_matricesConst[matricesIndices[0]], 64); //initialPermutation(input);
	permuteMatrixCuda(sharedKey, sharedCopy, &d_matricesConst[matricesIndices[1]], 56); // PC1 of key

	for (int i = 0; i < 16; i++)
	{
		// Preserving L,R.
		// preserve right side, R. (Result[63:32] = Input[31:0])
		sharedResult[threadIdx.x] = (threadIndex >= 32) ? sharedInput[threadIdx.x - 32] : 0;

		// preserve left side, L. (Left[31:0] = Input[63:32])
		sharedLeft[threadIdx.x] = (threadIndex < 32) ? sharedInput[threadIdx.x + 32] : 0;
		// Round key
		// Shift key to the right for the next iteration
		leftCircularShiftCuda(sharedKey, sharedCopy, d_matricesConst[matricesIndices[6]+i]);

		// Preserve the current shifted key (sharedKey) for the next iteration.
		sharedRoundkey[threadIdx.x] = sharedKey[threadIdx.x];
		__syncthreads();

		// Permutation PC2 of the roundKey
		permuteMatrixCuda(sharedRoundkey, sharedCopy, &d_matricesConst[matricesIndices[2]], 48);//roundKeyPermutation(permutedRoundKey);

		// Expansion permutation of input's right side (R).
		permuteMatrixCuda(sharedInput, sharedCopy, &d_matricesConst[matricesIndices[3]], 48);//expandPermutation(input); // 48 bits

		// XOR with permuted round key
		sharedInput[threadIdx.x] = sharedInput[threadIdx.x] ^ sharedRoundkey[threadIdx.x];
		__syncthreads();

		// Substitution S-boxes
		substituteCuda(sharedInput, sharedX, sharedY, d_SBoxesConst);

		// "P-matrix" permutation i.e. mix/shuffle
		permuteMatrixCuda(sharedInput, sharedCopy, &d_matricesConst[matricesIndices[4]], 32);// mixPermutation(input);

		// XOR with preserved left side
		if (threadIndex < 32)
		{
			sharedResult[threadIdx.x] = sharedLeft[threadIdx.x] ^ sharedInput[threadIdx.x];
		}
		__syncthreads();

		// End of loop
		sharedInput[threadIdx.x] = sharedResult[threadIdx.x];
		__syncthreads();
	}

	swapLRCuda(sharedResult, sharedCopy);
	permuteMatrixCuda(sharedResult, sharedCopy, &d_matricesConst[matricesIndices[5]], 64);//reverseInitialPermutation(result);

	if (threadIndex == 0)
	{
		result = 0;
		for (int i = 0; i < 64; i++)
		{
			result <<= 1;
			result += sharedResult[63 - i + (threadSetIndex << 6)] & 1;
		}
		results[(blockIdx.x << 1) + threadSetIndex] = result;
	}
	__syncthreads();
}
//
__global__ void DecryptDESCuda(uint64_t* encryptions, uint64_t* keys, uint64_t* results, const unsigned char* d_matricesConst, const unsigned char* d_SBoxesConst)
{
	// Kernel iterations shared memory
	__shared__ unsigned char sharedInput[128];
	__shared__ unsigned char sharedLeft[128];
	__shared__ unsigned char sharedResult[128];
	__shared__ unsigned char sharedKey[128];
	__shared__ unsigned char sharedRoundkey[128];
	__shared__ uint64_t result; // setting alias for encryption

	// General shared array. Typically for copying input. Used in the following functions: permuteMatrixCuda, swapLRCuda, leftCircularShiftCuda, rightCircularShiftCuda
	__shared__ unsigned char sharedCopy[128];
	// Special arrays for 'subsituteCuda' function:
	__shared__ uint16_t sharedX[16];
	__shared__ uint16_t sharedY[16];

	uint64_t input;
	uint64_t shiftedKey;
	const int matricesIndices[7] = { 0, 64,120,168,216,248,312 };

	int threadSetIndex = threadIdx.x >> 6; // divide by 64=2^6
	int threadIndex = (threadIdx.x >= 64) ? (threadIdx.x - 64) : threadIdx.x;
	// Initializations
	input = encryptions[(blockIdx.x << 1) + threadSetIndex];
	shiftedKey = keys[(blockIdx.x << 1) + threadSetIndex];
	sharedInput[threadIdx.x] = 0;
	sharedLeft[threadIdx.x] = 0;
	sharedResult[threadIdx.x] = 0;
	sharedKey[threadIdx.x] = 0;
	sharedRoundkey[threadIdx.x] = 0;
	sharedCopy[threadIdx.x] = 0;
	if (threadIndex < 8)
	{
		sharedX[threadIdx.x] = 0;
		sharedY[threadIdx.x] = 0;
	}
	if (threadIndex == 0)
	{
		result = 0;
	}
	__syncthreads();

	// Initial operations 
	// The 64 bits of message,key (uint64_t) are converted into 64 bytes (unsigned char) so that they are easily parallelizable. 
	sharedInput[threadIdx.x] = (input >> threadIndex) & 1;
	sharedKey[threadIdx.x] = (shiftedKey >> threadIndex) & 1;
	__syncthreads();

	// Initial permutation parallelized
	permuteMatrixCuda(sharedInput, sharedCopy, &d_matricesConst[matricesIndices[0]], 64); //initialPermutation(input);
	permuteMatrixCuda(sharedKey, sharedCopy, &d_matricesConst[matricesIndices[1]], 56); // PC1 of key

	__syncthreads();
	for (int i = 0; i < 16; i++)
	{
		// Preserving L,R.
		// preserve right side, R. (Result[63:32] = Input[31:0])
		sharedResult[threadIdx.x] = (threadIndex >= 32) ? sharedInput[threadIdx.x - 32] : 0;

		// preserve left side, L. (Left[31:0] = Input[63:32])
		sharedLeft[threadIdx.x] = (threadIndex < 32) ? sharedInput[threadIdx.x + 32] : 0;
		__syncthreads();

		// Round key
		// Preserve the current shifted key (sharedKey) for the next iteration.
		sharedRoundkey[threadIdx.x] = sharedKey[threadIdx.x];
		__syncthreads();

		// Permutation PC2 of the roundKey
		permuteMatrixCuda(sharedRoundkey, sharedCopy, &d_matricesConst[matricesIndices[2]], 48);//roundKeyPermutation(permutedRoundKey);

		// Shift key to the right for the next iteration
		rightCircularShiftCuda(sharedKey, sharedCopy, d_matricesConst[matricesIndices[6] + 15-i]);

		// Expansion permutation of input's right side (R).
		permuteMatrixCuda(sharedInput, sharedCopy, &d_matricesConst[matricesIndices[3]], 48);//expandPermutation(input); // 48 bits

		// XOR with permuted round key
		sharedInput[threadIdx.x] = sharedInput[threadIdx.x] ^ sharedRoundkey[threadIdx.x];
		__syncthreads();

		// Substitution S-boxes
		substituteCuda(sharedInput, sharedX, sharedY, d_SBoxesConst);

		// "P-matrix" permutation i.e. mix/shuffle
		permuteMatrixCuda(sharedInput, sharedCopy, &d_matricesConst[matricesIndices[4]], 32);// mixPermutation(input);

		// XOR with preserved left side
		if (threadIndex < 32)
		{
			sharedResult[threadIdx.x] = sharedLeft[threadIdx.x] ^ sharedInput[threadIdx.x];
		}
		__syncthreads();

		// End of loop
		sharedInput[threadIdx.x] = sharedResult[threadIdx.x];
		__syncthreads();
	}

	swapLRCuda(sharedResult, sharedCopy);
	permuteMatrixCuda(sharedResult, sharedCopy, &d_matricesConst[matricesIndices[5]], 64);//reverseInitialPermutation(result);

	if (threadIndex == 0)
	{
		result = 0;
		for (int i = 0; i < 64; i++)
		{
			result <<= 1;
			result += sharedResult[63 - i + (threadSetIndex << 6)] & 1;
		}
		results[(blockIdx.x << 1) + threadSetIndex] = result;
	}
	__syncthreads();
}

__device__ void permuteMatrixCuda(unsigned char* input, unsigned char* sharedCopy, const unsigned char* P, unsigned int size)
{
	int threadIndex = (threadIdx.x >= 64) ? (threadIdx.x - 64) : threadIdx.x; // divide by 64=2^6
	sharedCopy[threadIdx.x] = input[threadIdx.x];
	__syncthreads();

	// if thread is bigger than the alloted permutation size, make slot equal to 0.
	// Note (threadIdx.x%size) is used in case of a memory violation. But, this precaution is rendered unnecessary by the ternary operator.
	unsigned char bit;
	bit = (threadIndex >= size) ? 0 : (sharedCopy[P[threadIndex%size] - 1] & 1);
	input[threadIdx.x] = bit;
	__syncthreads();
}

__device__ void leftCircularShiftCuda(unsigned char* input, unsigned char* sharedCopy, uint8_t times)
{
	int threadSetIndex = threadIdx.x >> 6;
	int threadIndex = (threadIdx.x >= 64) ? (threadIdx.x - 64) : threadIdx.x;
	
	// copying the key
	sharedCopy[threadIdx.x] = input[threadIdx.x];
	__syncthreads();

	// set offset to determine left and right (L,R) sides of key.
	int offset = 28 * (threadIndex / 28);

	int index = offset + (threadIndex + times) % 28;

	// accounting for edge case with 64 bits.
	// Note shifting is not necessary here, as we do not care about the last 8 bits. 
	index = (index >= 56) ? (offset+index%8) : index;

	// Finally applying the shift
	input[index + (threadSetIndex << 6)] = sharedCopy[threadIdx.x];
	__syncthreads();
}

// Note: maximum of 28 shifts at call of function
__device__ void rightCircularShiftCuda(unsigned char* input, unsigned char* sharedCopy, uint8_t times)
{
	int threadSetIndex = threadIdx.x >> 6;
	int threadIndex = (threadIdx.x >= 64) ? (threadIdx.x - 64) : threadIdx.x;
	// copying the key
	sharedCopy[threadIdx.x] = input[threadIdx.x];
	__syncthreads();

	// set offset to determine left and right (L,R) sides of key.
	int offset = 28 * (threadIndex / 28);

	int index = offset + (threadIndex + 28 - times) % 28;

	// accounting for edge case with 64 bits.
	// Note shifting is not necessary here, as we do not care about the last 8 bits. 
	index = (index >= 56) ? (offset + index % 8) : index;

	// Finally applying the shift
	input[index + (threadSetIndex << 6)] = sharedCopy[threadIdx.x];
	__syncthreads();
}

__device__ void substituteCuda(unsigned char* input, uint16_t* sharedX, uint16_t* sharedY, const unsigned char* d_SBoxesConst)
{
	// 16 inputs (8 x,y pairs) and 8 outputs - 8 extractions from SBox. 
	// 64 threads will allow for 8 simulataneous extractions.

	// Thus, 16 threads will suffice to calculate x,y pairs.
	// 8 threads for each of x and y.

	int threadSetIndex = threadIdx.x >> 6;
	int threadIndex = (threadIdx.x >= 64) ? (threadIdx.x - 64) : threadIdx.x;
	int tid = threadIdx.x;
	int setIndex, index, threadPos;
	uint8_t x = 0;
	uint8_t y = 0;
	unsigned char byte, bit;

	// Y calculation
	// Threads 0 -> 7 work here - First warp
	if (threadIndex < 8)
	{
		// y = b5,b0 then b11,b6, b17,b12 ... b47,b42 i.e. tid*6 + 5, tid*6
		y = (input[(threadSetIndex << 6) + threadIndex * 6 + 5]) << 1;
		y |= input[(threadSetIndex << 6) + threadIndex * 6];
		sharedY[threadIndex + (threadSetIndex << 3)] = y;
	}

	// X calculation 
	// Threads 32 -> 39 work here - Second warp
	if (threadIndex >= 32 && threadIndex < 40)
	{
		// x = b4,b3,b2,b1 then b10,...,b7 i.e. tid * 6 + 4, ..., tid * 6 + 1
		// note we reduced tid by 8, as we work with threads 8->15.
		// i.e. (x's index) * 6 + 4, ..., (x's index) * 6 + 1
		index = threadIndex - 32;
		for (int i = 0; i < 4; i++)
		{
			x |= input[index * 6 + (i + 1) + (threadSetIndex << 6)] << i;
		}
		sharedX[index + (threadSetIndex << 3)] = x;
	}

	// Warps are joined here.
	__syncthreads(); 

	// Extract Sbox output and place it into 'input'
	if (threadIndex < 32)
	{
		setIndex = threadIndex / 4;
		threadPos = threadIndex % 4;
		byte = d_SBoxesConst[(setIndex << 6) + (sharedY[setIndex + (threadSetIndex << 3)] << 4) + (sharedX[setIndex + (threadSetIndex << 3)]) ];
		byte >>= threadPos;
		bit = byte & 1;

		input[tid] = byte & 1;
	}

	// Wipe the last 32 bits of input.
	if (threadIndex >= 32)
	{
		input[tid] = 0;
	}
	__syncthreads();

}

__device__ void swapLRCuda(unsigned char* input, unsigned char* sharedCopy) // Swap left (32 bit) and right (32 bit) parts of the 64 bit input.
{
	sharedCopy[threadIdx.x] = input[threadIdx.x];
	__syncthreads();

	input[threadIdx.x] = sharedCopy[(threadIdx.x + 32)%64 + ((threadIdx.x>=64) << 6)];
	__syncthreads();
}