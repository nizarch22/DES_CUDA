#include "hip/hip_runtime.h"
#include <cstdlib>
// External
#include "hip/hip_runtime.h"
#include ""
#include "DES_CUDA.cuh"


// Definitions
#define NUM_THREADS 128
#define NUM_TESTS 300

__global__ void EncryptDESCuda(uint64_t* messages, uint64_t* keys, unsigned char* matrices, unsigned char* sboxes, uint64_t* results)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint64_t result; // setting alias for encryption

	uint64_t input = messages[tid];
	uint64_t shiftedKey = keys[tid];
	uint64_t permutedRoundKey;
	uint64_t left; // last 32 bits of plaintext/input to algorithm are preserved in this variable 

	// load matrices
	// essential variables
	unsigned char* cIP, * cPC1, * cPC2, * cE, * cPMatrix, * cIPInverse, * cLCS;
	int matricesSizes[7] = { 64,56,48,48,32,64,16 };

	// loading matrices process
	unsigned char* temp = matrices;
	cIP = temp; temp += matricesSizes[0];
	cPC1 = temp; temp += matricesSizes[1];
	cPC2 = temp; temp += matricesSizes[2];
	cE = temp; temp += matricesSizes[3];
	cPMatrix = temp; temp += matricesSizes[4];
	cIPInverse = temp; temp += matricesSizes[5];
	cLCS = temp;

	// Initial operations 
	permuteMatrixCuda(input, cIP, 64); //initialPermutation(input);
	permuteMatrixCuda(shiftedKey, cPC1, 56); // PC1 of key
	for (int i = 0; i < 16; i++)
	{
		// Preserving L,R.
		// preserve right side (Result[63:32] = Input[31:0])
		result = input;
		result <<= 32;
		// preserve left side
		left = input >> 32;

		// Round key
		generateShiftedKeyCuda(i, shiftedKey, cLCS);
		permutedRoundKey = shiftedKey;
		permuteMatrixCuda(permutedRoundKey, cPC2, 48);//roundKeyPermutation(permutedRoundKey);

		// Expansion permutation
		permuteMatrixCuda(input, cE, 48);//expandPermutation(input); // 48 bits

		// XOR with permuted round key
		input ^= permutedRoundKey;

		// Substitution S-boxes
		substituteCuda(input, sboxes); // 32 bits

		// "P-matrix" permutation i.e. mix/shuffle
		permuteMatrixCuda(input, cPMatrix, 32);// mixPermutation(input);

		// XOR with preserved left side
		result += left ^ input; // Result[31:0] = L XOR f[31:0];

		// End of loop
		input = result;
	}

	swapLRCuda(result);
	permuteMatrixCuda(result, cIPInverse, 64);//reverseInitialPermutation(result);
	results[tid] = result;
	// debug final point
}
__global__ void DecryptDESCuda(uint64_t* messages, uint64_t* keys, unsigned char* matrices, unsigned char* sboxes, uint64_t* results)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint64_t result; // setting alias for encryption

	uint64_t input = messages[tid];
	uint64_t shiftedKey = keys[tid];
	uint64_t permutedRoundKey;
	uint64_t left; // last 32 bits of plaintext/input to algorithm are preserved in this variable 

	// load matrices
	// essential variables
	unsigned char* cIP, * cPC1, * cPC2, * cE, * cPMatrix, * cIPInverse, * cLCS;
	int matricesSizes[7] = { 64,56,48,48,32,64,16 };

	// loading matrices process
	int offset = 0;
	unsigned char* temp = matrices;
	cIP = temp; temp += matricesSizes[0];
	cPC1 = temp; temp += matricesSizes[1];
	cPC2 = temp; temp += matricesSizes[2];
	cE = temp; temp += matricesSizes[3];
	cPMatrix = temp; temp += matricesSizes[4];
	cIPInverse = temp; temp += matricesSizes[5];
	cLCS = temp;


	// Initial operations 
	permuteMatrixCuda(input, cIP, 64); //initialPermutation(input);
	permuteMatrixCuda(shiftedKey, cPC1, 56); // PC1 of key
	fullShiftLCSCuda(shiftedKey);


	for (int i = 0; i < 16; i++)
	{
		// Preserving L,R.
		// preserve right side (Result[63:32] = Input[31:0])
		result = input;
		result <<= 32;
		// preserve left side
		left = input >> 32;

		// Round key
		permutedRoundKey = shiftedKey;
		permuteMatrixCuda(permutedRoundKey, cPC2, 48);//roundKeyPermutation(permutedRoundKey);
		generateShiftedKeyCuda(i, shiftedKey, cLCS);

		// Expansion permutation
		permuteMatrixCuda(input, cE, 48);//expandPermutation(input); // 48 bits

		// XOR with permuted round key
		input ^= permutedRoundKey;

		// Substitution S-boxes
		substituteCuda(input, sboxes); // 32 bits

		// "P-matrix" permutation i.e. mix/shuffle
		permuteMatrixCuda(input, cPMatrix, 32);// mixPermutation(input);

		// XOR with preserved left side
		result += left ^ input; // Result[31:0] = L XOR f[31:0];

		// End of loop
		input = result;
	}

	swapLRCuda(result);
	permuteMatrixCuda(result, cIPInverse, 64);//reverseInitialPermutation(result);
	results[tid] = result;
}

__global__ void EncryptDESCudaDebug(uint64_t* messages, uint64_t* keys, unsigned char* matrices, unsigned char* sboxes, uint64_t* results, uint64_t* debug, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint64_t result; // setting alias for encryption

	uint64_t input = messages[tid];
	uint64_t shiftedKey = keys[tid];
	uint64_t permutedRoundKey;
	uint64_t left; // last 32 bits of plaintext/input to algorithm are preserved in this variable 
	
	// load matrices
	// essential variables
	unsigned char* cIP, * cPC1, * cPC2, * cE, * cPMatrix, * cIPInverse, * cLCS;
	int matricesSizes[7] = { 64,56,48,48,32,64,16 };

	// loading matrices process
	unsigned char* temp = matrices;
	cIP = temp; temp += matricesSizes[0];
	cPC1 = temp; temp += matricesSizes[1];
	cPC2 = temp; temp += matricesSizes[2];
	cE = temp; temp += matricesSizes[3];
	cPMatrix = temp; temp += matricesSizes[4];
	cIPInverse = temp; temp += matricesSizes[5];
	cLCS = temp;

	// Initial operations 
	permuteMatrixCuda(input, cIP, 64); //initialPermutation(input);
	debug[0 + tid * n] = input;
	debug[1 + tid * n] = shiftedKey;
	permuteMatrixCuda(shiftedKey, cPC1, 56); // PC1 of key
	debug[2 + tid * n] = shiftedKey;
	for (int i = 0; i < 16; i++)
	{
		// Preserving L,R.
		// preserve right side (Result[63:32] = Input[31:0])
		result = input;
		result <<= 32;
		// preserve left side
		left = input >> 32;

		// Round key
		generateShiftedKeyCuda(i, shiftedKey, cLCS);
		debug[3 + tid * n] = shiftedKey;
		permutedRoundKey = shiftedKey;
		permuteMatrixCuda(permutedRoundKey, cPC2, 48);//roundKeyPermutation(permutedRoundKey);
		debug[4 + tid * n] = permutedRoundKey;

		// Expansion permutation
		permuteMatrixCuda(input, cE, 48);//expandPermutation(input); // 48 bits
		debug[5 + tid * n] = input;

		// XOR with permuted round key
		input ^= permutedRoundKey;
		debug[6 + tid * n] = input;
		// Substitution S-boxes
		substituteCuda(input, sboxes); // 32 bits
		debug[7 + tid * n] = input;

		// "P-matrix" permutation i.e. mix/shuffle
		permuteMatrixCuda(input, cPMatrix, 32);// mixPermutation(input);
		debug[8 + tid * n] = input;

		// XOR with preserved left side
		result += left ^ input; // Result[31:0] = L XOR f[31:0];

		// End of loop
		input = result;
	}

	swapLRCuda(result);
	debug[9 + tid * n] = result;
	permuteMatrixCuda(result, cIPInverse, 64);//reverseInitialPermutation(result);
	debug[10 + tid * n] = result;
	results[tid] = result;
	// debug final point
	debug[11 + tid * n] = messages[tid];
}



__device__ void permuteMatrixCuda(uint64_t& input, const unsigned char* P, const unsigned int size)
{
	uint64_t output = 0;
	uint64_t bit;

	for (int i = 0; i < size; i++)
	{
		bit = (input >> (P[i] - 1)) & 1;
		output += bit << i;
	}
	input = output;
}
__device__ void generateShiftedKeyCuda(const int& index, uint64_t& roundKey, unsigned char* cLCS)
{
	uint32_t left, right;
	uint64_t mask28Bits = 268435455; // covers first 28 bits

	// getting left and right sides
	right = roundKey & mask28Bits;
	mask28Bits <<= 28;
	mask28Bits = roundKey & mask28Bits;
	left = mask28Bits >> 28;

	// circular shifts
	leftCircularShiftCuda(left, cLCS[index]);
	leftCircularShiftCuda(right, cLCS[index]);

	// copying left and right shifted keys to roundKey.
	roundKey = left;
	roundKey <<= 28;
	roundKey += right;
}
__device__ void generateReverseShiftedKeyCuda(const int& index, uint64_t& roundKey, unsigned char* cLCS)
{
	uint32_t left, right;
	uint64_t mask28Bits = 268435455; // covers first 28 bits

	// getting left and right sides
	right = roundKey & mask28Bits;
	mask28Bits <<= 28;
	mask28Bits = roundKey & mask28Bits;
	left = mask28Bits >> 28;

	// circular shifts
	rightCircularShiftCuda(left, cLCS[15 - index]);
	rightCircularShiftCuda(right, cLCS[15 - index]);

	// copying left and right shifted keys to roundKey.
	roundKey = left;
	roundKey <<= 28;
	roundKey += right;
}
__device__ void leftCircularShiftCuda(uint32_t& input, uint8_t times)
{
	uint32_t mask28thBit = 1 << 27; // 28th bit
	uint32_t mask28Bits = 268435455; // covers first 28 bits

	uint8_t bit;
	for (int i = 0; i < times; i++)
	{
		bit = (input & mask28thBit) >> 27;
		input <<= 1;
		input += bit;
	}
	input = input & mask28Bits;
}

__device__ void rightCircularShiftCuda(uint32_t& input, uint8_t times)
{
	uint32_t bit28th = 1 << 27; // 28th bit
	uint32_t mask1stBit = 1; // 28th bit
	uint32_t mask28Bits = 268435455; // covers first 28 bits

	uint32_t bit;
	for (int i = 0; i < times; i++)
	{
		bit = (input & mask1stBit);
		input >>= 1;
		input += bit * bit28th;
	}
	input = input & mask28Bits;
}

__device__ void substituteCuda(uint64_t& input, unsigned char* sboxes)
{
	uint64_t result = 0; uint64_t temp;
	uint8_t y, x;
	uint8_t in;

	uint64_t mask = 63;
	uint8_t maskY1, maskY2, maskX;
	maskY1 = 1;
	maskY2 = 32;
	maskX = 30;
	for (int i = 0; i < 8; i++)
	{
		// getting x,y coordinates for Sbox
		in = input & mask;
		x = (in & maskX) >> 1;
		y = (in & maskY2) >> 4;
		y += in & maskY1;

		// Substitution 
		temp = sboxes[i*64+y*16+x];
		result += temp << (4 * i);

		// next bits
		input >>= 6;
	}
	input = result;
}
__device__ void swapLRCuda(uint64_t& input) // Swap left (32 bit) and right (32 bit) parts of the 64 bit input.
{
	uint64_t temp = input;
	// containing left side 
	temp >>= 32;

	// right side moved to left
	input <<= 32;

	// left side moved to right
	input += temp;
}

// Preemptively shifting all keys using LCS matrix.
__device__ void fullShiftLCSCuda(uint64_t& roundKey)
{
	uint32_t left, right;
	uint64_t mask28Bits = 268435455; // covers first 28 bits

	// getting left and right sides
	right = roundKey & mask28Bits;
	mask28Bits <<= 28;
	mask28Bits = roundKey & mask28Bits;
	left = mask28Bits >> 28;
	
	// circular shifts
	leftCircularShiftCuda(left, 28);
	leftCircularShiftCuda(right, 28);

	// copying left and right shifted keys to roundKey.
	roundKey = left;
	roundKey <<= 28;
	roundKey += right;
}