#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "DES.h"
#include <stdio.h>
#include <iostream>

__global__ void EncryptDESCuda(uint64_t* messages, uint64_t* keys, unsigned char* matrices, unsigned char* sboxes, uint64_t* results);
__global__ void EncryptDESCudaDebug(uint64_t* messages, uint64_t* keys, unsigned char* matrices, unsigned char* sboxes, uint64_t* results, uint64_t* debug, int n);
void EncryptDESDebug(const uint64_t& plaintext, const uint64_t& key, uint64_t& encryption, uint64_t* debug);
void printCharMatrix(unsigned char* matrix, int y, int x);
int main()
{
    // kernel parameters
    const int numThreads = 512;
    const int numMessages = 524288;// 4MB - 10x speedup//33554432; // 268MB - 223 speedup!
    const int numBlocks = (numMessages + numThreads - 1) / numThreads;

    // size parameters
    int bytesMessages = sizeof(uint64_t) * numMessages;
    int bytesKeys = sizeof(uint64_t) * numMessages;

    // kernel argument prep stage
    // 
    // prep matrices, sboxes
    unsigned char* d_SBoxes, * d_matrices;
    unsigned char* matrices[7] = {IP,PC1,PC2, E, PMatrix,IPInverse, LCS};
    int matricesSizes[7] = { 64,56,48,48,32,64,16 };
    // prep keys, messages, encryptions, decryptions
    uint64_t* d_messages, * d_keys;
    uint64_t* messages = (uint64_t*)malloc(bytesMessages);
    uint64_t* keys = (uint64_t*)malloc(bytesKeys);
    for (int i = 0; i < numMessages; i++)
    {
        messages[i] = (((uint64_t)rand()) << 32) | rand();
        keys[i] = (((uint64_t)rand()) << 32) | rand();
    }
    
    // prep results
    uint64_t* d_resultsEncryption, * d_resultsDecryption;
    uint64_t* resultsEncryption = (uint64_t*)malloc(bytesMessages);
    uint64_t* resultsDecryption = (uint64_t*)malloc(bytesMessages);
    // CPU-run DES Results
    uint64_t* encryptions = (uint64_t*)malloc(bytesMessages);
    uint64_t* decryptions = (uint64_t*)malloc(bytesMessages);

    int startTimeAlloc = clock();
    // cuda allocate memory - matrices, sboxes
    const int matricesSize = 328;
    const int sboxesSize= 512;
    hipMalloc(&d_matrices, matricesSize);
    hipMalloc(&d_SBoxes, sboxesSize);
    // cuda allocate memory - messages, keys
    hipMalloc(&d_messages, bytesMessages);
    hipMalloc(&d_keys, bytesKeys);
    // cuda allocate memory - results
    hipMalloc(&d_resultsEncryption, bytesMessages);
    hipMalloc(&d_resultsDecryption, bytesMessages);

    int startTime = clock();
    // cuda copy memory - matrices, sboxes
    hipMemcpy(d_SBoxes, &SBoxes[0][0], 64*8, hipMemcpyHostToDevice);
    int offset = 0;
    for (int i = 0; i < 7; i++)
    {
        hipMemcpy(d_matrices + offset, &matrices[i][0], matricesSizes[i], hipMemcpyHostToDevice);
        offset += matricesSizes[i];
    }
    // cuda copy memory - messages, keys
    hipMemcpy(d_messages, messages, bytesMessages, hipMemcpyHostToDevice);
    hipMemcpy(d_keys, keys, bytesKeys, hipMemcpyHostToDevice);

    // Encryption cuda stage
    //
    //
    //EncryptDESCuda<<<numBlocks,numThreads>>>(d_messages, d_keys, d_matrices, d_SBoxes, d_resultsEncryption);

    //// results retrieval stage
    ////
    ////
    //hipMemcpy(resultsEncryption, d_resultsEncryption, bytesMessages, hipMemcpyDeviceToHost);
    //hipDeviceSynchronize(); // remove?
    //for (int i = 0; i < numMessages; i++)
    //{
    //    //printMatrix(resultsEncryption[i], 8, 8);
    //}
    //
    //// CPU validate encryption results stage
    ////
    ////
    //int bSame = 1;
    //uint64_t message, key, encryption;
    //for (int i = 0; i < numMessages; i++)
    //{
    //    message = messages[i]; key = keys[i];
    //    EncryptDES(messages[i], keys[i], encryption);
    //    bSame &= encryption == resultsEncryption[i];
    //    if (!bSame)
    //    {
    //        //std::cout << "Operation failed!\n";
    //        //printMatrix(encryption, 8, 8);
    //    }
    //}

    // Debugging stage
    // 
    // 
    EncryptDESCuda << <numBlocks, numThreads >> > (d_messages, d_keys, d_matrices, d_SBoxes, d_resultsEncryption);
    // copy result from cuda
    hipMemcpy(resultsEncryption, d_resultsEncryption, bytesMessages, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    int endTime = clock();
    int CUDATime = endTime - startTimeAlloc;
    int CUDATimeCopy = endTime - startTime;
    startTime = clock();
    for (int i = 0; i < numMessages; i++)
    {
        EncryptDES(messages[i], keys[i], encryptions[i]);
        //DecryptDES(encryptions[i], keys[i], decryptions[i]);
    }
    endTime = clock();
    int CPUTime = endTime - startTime;

    std::cout << "CUDA Debug results:\n";

    std::cout << "Total time to allocate memory + copy memory back and forth:\n";
    std::cout << "GPU: " << CUDATime << "ms\n";
    std::cout << "CPU: " << CPUTime << "ms\n";
    std::cout << "GPU - only since copying: " << CUDATimeCopy << "ms\n";
    double speedup = (float)CPUTime / CUDATime;
    double speedupCopy = (float)CPUTime / CUDATimeCopy;
    std::cout << "Total speedup: " << speedup << "\n";
    std::cout << "speedup without counting allocation: " << speedupCopy << "\n";

    // confirming that indeed we have the correction results
    bool bEqual = 1;
    for (int i = 0; i < numMessages; i++)
    {
        bEqual &= (encryptions[i] == resultsEncryption[i]);
    }
    if (bEqual)
    {
        std::cout << "Success!\n";
    }

     //Decryption cuda stage
    
    
    return 0;

   // return 0;
   // // break here.
   // int* c = (int*)malloc(bytes+sizeof(int));
   // int* d_c;
   // hipMalloc(&d_c, bytes+4);
   // for (int i = 0; i < arraySize; i++)
   // {
   //     c[i] = 1000;
   // }
   // hipMemcpy(d_c, c, (bytes), hipMemcpyHostToDevice);

   // // Add vectors in parallel.
   // //EncryptDESCuda <<<1, 1>>>(d_c);
   // //if (cudaStatus != hipSuccess) {
   // //    fprintf(stderr, "addWithCuda failed!");
   // //    return 1;
   // //}
   //hipMemcpy(c, d_c, bytes+4, hipMemcpyDeviceToHost);
   //for (int i = 0; i < arraySize+1; i++)
   //{
   //    std::cout << c[i] << ",";
   //}
   //std::cout << "\n";
   // printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        //c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}



void printCharMatrix(unsigned char* matrix, int y, int x)
{
    //bool bit;
    //bool mask = 1;
    for (int i = 0; i < y; i++)
    {
        for (int j = 0; j < x; j++)
        {

            //bit = matrix & mask;
            std::cout << matrix[i*y+j] << ",";
            //matrix >>= 1;
        }
        std::cout << "\n";
    }
    std::cout << "Matrix printed.\n";
}

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void lengthTest(unsigned char* matrices, unsigned char* sboxes, unsigned char* results)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    results[tid] = sboxes[tid];
}

__global__ void cudaTest(unsigned char* a, unsigned char* b)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    b[tid] = tid;
}

// major helper functions
//__device__ void permuteMatrixCuda()
//{
//
//}
//__device__ void substituteCuda()
//{
//
//}
//__device__ void leftCircularShiftCuda()
//{
//
//}
//__device__ void 

