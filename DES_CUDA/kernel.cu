#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "DES.h"
#include <stdio.h>
#include <iostream>

__global__ void EncryptDESCuda(uint64_t* messages, uint64_t* keys, unsigned char* matrices, unsigned char* sboxes, uint64_t* results);
__global__ void EncryptDESCudaDebug(uint64_t* messages, uint64_t* keys, unsigned char* matrices, unsigned char* sboxes, uint64_t* results, uint64_t* debug, int n);
void EncryptDESDebug(const uint64_t& plaintext, const uint64_t& key, uint64_t& encryption, uint64_t* debug);
void printCharMatrix(unsigned char* matrix, int y, int x);
int main()
{
    // kernel parameters
    const int numThreads = 128;
    const int numMessages = 1024;
    const int numBlocks = (numMessages + numThreads - 1) / numThreads;

    // size parameters
    int bytesMessages = sizeof(uint64_t) * numMessages;
    int bytesKeys = sizeof(uint64_t) * numMessages;

    // kernel argument prep stage
    // 
    // prep matrices, sboxes
    unsigned char* d_SBoxes, * d_matrices;
    unsigned char* matrices[7] = {IP,PC1,PC2, E, PMatrix,IPInverse, LCS};
    int matricesSizes[7] = { 64,56,48,48,32,64,16 };
    // prep keys, messages
    uint64_t* d_messages, * d_keys;
    uint64_t* messages = (uint64_t*)malloc(bytesMessages);
    uint64_t* keys = (uint64_t*)malloc(bytesKeys);
    for (int i = 0; i < numMessages; i++)
    {
        messages[i] = (((uint64_t)rand()) << 32) | rand();
        keys[i] = (((uint64_t)rand()) << 32) | rand();
    }
    // prep results
    uint64_t* d_resultsEncryption, * d_resultsDecryption;
    uint64_t* resultsEncryption = (uint64_t*)malloc(bytesMessages);
    uint64_t* resultsDecryption = (uint64_t*)malloc(bytesMessages);

    // cuda allocate memory - matrices, sboxes
    hipMalloc(&d_matrices, 328);
    hipMalloc(&d_SBoxes, 8*64);
    // cuda allocate memory - messages, keys
    hipMalloc(&d_messages, bytesMessages);
    hipMalloc(&d_keys, bytesKeys);
    // cuda allocate memory - results
    hipMalloc(&d_resultsEncryption, bytesMessages);
    hipMalloc(&d_resultsDecryption, bytesMessages);

    // copy memory - matrices, sboxes
    hipMemcpy(d_SBoxes, &SBoxes[0][0], 64*8, hipMemcpyHostToDevice);
    int offset = 0;
    for (int i = 0; i < 7; i++)
    {
        hipMemcpy(d_matrices + offset, &matrices[i][0], matricesSizes[i], hipMemcpyHostToDevice);
        offset += matricesSizes[i];
    }
    // copy memory - messages, keys
    hipMemcpy(d_messages, messages, bytesMessages, hipMemcpyHostToDevice);
    hipMemcpy(d_keys, keys, bytesKeys, hipMemcpyHostToDevice);

    // Encryption cuda stage
    //
    //
    EncryptDESCuda<<<numBlocks,numThreads>>>(d_messages, d_keys, d_matrices, d_SBoxes, d_resultsEncryption);

    // results retrieval stage
    //
    //
    hipMemcpy(resultsEncryption, d_resultsEncryption, bytesMessages, hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); // remove?
    for (int i = 0; i < numMessages; i++)
    {
        //printMatrix(resultsEncryption[i], 8, 8);
    }
    
    // CPU validate encryption results stage
    //
    //
    int bSame = 1;
    uint64_t message, key, encryption;
    for (int i = 0; i < numMessages; i++)
    {
        message = messages[i]; key = keys[i];
        EncryptDES(messages[i], keys[i], encryption);
        bSame &= encryption == resultsEncryption[i];
        if (!bSame)
        {
            //std::cout << "Operation failed!\n";
            //printMatrix(encryption, 8, 8);
        }
    }

    // Debugging stage
    // 
    // 
    const int numDebugs = 12;
    const int numTotalDebugs = numDebugs * numMessages;
    const int sizeDebug = (numTotalDebugs) * sizeof(uint64_t);
    uint64_t arrDebug[numTotalDebugs];
    uint64_t cudaArrDebug[numTotalDebugs];
    // malloc
    uint64_t* d_arrDebug;
    hipMalloc(&d_arrDebug, sizeDebug);
    
    EncryptDESCudaDebug << <numBlocks, numThreads >> > (d_messages, d_keys, d_matrices, d_SBoxes, d_resultsEncryption, d_arrDebug, numDebugs);
    // copy result from cuda
    hipMemcpy(cudaArrDebug, d_arrDebug, sizeDebug, hipMemcpyDeviceToHost);
    hipMemcpy(resultsEncryption, d_resultsEncryption, bytesMessages, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    std::cout << "CUDA Debug results:\n";
    
    bool bEqual = 1;
    for (int j = 0; j < numMessages; j++)
    {
        EncryptDESDebug(messages[j], keys[j], encryption, arrDebug);
        for (int i = 0; i < numDebugs-1; i++)
        {
            if (cudaArrDebug[i+j*numDebugs] != arrDebug[i])
            {
                std::cout << "Fail happened at " << i << " on message number " << j << "\n";
                printMatrix(cudaArrDebug[i + j * numDebugs], 8, 8);
                printMatrix(arrDebug[i], 8, 8);
                // extra
                std::cout << "CUDA: " << cudaArrDebug[i + j * numDebugs] << "\nCPU: " << arrDebug[i] << "\n";
                std::cout << "Extras:\n";
                uint64_t extraVariable = messages[0];
                std::cout << "CUDA: " << cudaArrDebug[11 + j * numDebugs] << "\nCPU: " << extraVariable << "\n";
                //std::cout << cudaArrDebug[11+j*numDebugs] << "\n";
                bEqual = 1;
                break;
            }
        }
    }
    if (bEqual)
    {
        std::cout << "Success!\n";
    }
    for (int i = 0; i < numMessages; i++)
    {
        if (i % 20 == 0)
        {
            std::cout << "\n";
        }
        EncryptDESDebug(messages[i], keys[i], encryption, arrDebug);
        std::cout << (encryption == resultsEncryption[i]) << ",";

    }
    // Decryption cuda stage
    //
    //
    return 0;

   // return 0;
   // // break here.
   // int* c = (int*)malloc(bytes+sizeof(int));
   // int* d_c;
   // hipMalloc(&d_c, bytes+4);
   // for (int i = 0; i < arraySize; i++)
   // {
   //     c[i] = 1000;
   // }
   // hipMemcpy(d_c, c, (bytes), hipMemcpyHostToDevice);

   // // Add vectors in parallel.
   // //EncryptDESCuda <<<1, 1>>>(d_c);
   // //if (cudaStatus != hipSuccess) {
   // //    fprintf(stderr, "addWithCuda failed!");
   // //    return 1;
   // //}
   //hipMemcpy(c, d_c, bytes+4, hipMemcpyDeviceToHost);
   //for (int i = 0; i < arraySize+1; i++)
   //{
   //    std::cout << c[i] << ",";
   //}
   //std::cout << "\n";
   // printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        //c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}



void printCharMatrix(unsigned char* matrix, int y, int x)
{
    //bool bit;
    //bool mask = 1;
    for (int i = 0; i < y; i++)
    {
        for (int j = 0; j < x; j++)
        {

            //bit = matrix & mask;
            std::cout << matrix[i*y+j] << ",";
            //matrix >>= 1;
        }
        std::cout << "\n";
    }
    std::cout << "Matrix printed.\n";
}

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void lengthTest(unsigned char* matrices, unsigned char* sboxes, unsigned char* results)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    results[tid] = sboxes[tid];
}

__global__ void cudaTest(unsigned char* a, unsigned char* b)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    b[tid] = tid;
}

// major helper functions
//__device__ void permuteMatrixCuda()
//{
//
//}
//__device__ void substituteCuda()
//{
//
//}
//__device__ void leftCircularShiftCuda()
//{
//
//}
//__device__ void 


void testPrint()
{
    unsigned char* matrices[7] = { IP,PC1,PC2, E, PMatrix,IPInverse, LCS };
    int matricesSizes[7] = { 64,56,48,48,32,64,16 };
    // cuda call memory test
    unsigned char* arr; unsigned char* result;
    arr = (unsigned char*)malloc(328);
    result = (unsigned char*)malloc(328);

    // setup memory
    arr[0] = (char)244;
    arr[1] = (char)211;

    unsigned char* d_arr; unsigned char* d_result;
    hipMalloc(&d_arr, 328);
    hipMalloc(&d_result, 328);

    // copy arr memory
    int offset = 0;
    for (int i = 0; i < 7; i++)
    {
        hipMemcpy(d_arr + offset, &matrices[i][0], matricesSizes[i], hipMemcpyHostToDevice);
        offset += matricesSizes[i];
    }
    // run cuda
    cudaTest << <1, 328 >> > (d_arr, d_result);
    hipMemcpy(result, d_result, 328, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    offset = 0;
    std::cout << "Result:\n";
    for (int i = 0; i < 7; i++)
    {
        for (int j = 0; j < matricesSizes[i]; j++)
            std::cout << (int)result[offset + j] << ",";
        std::cout << "\n\n";
        offset += matricesSizes[i];
    }
}

void testPointerPrint()
{
    // doing testing
    unsigned char* ptrTest = &(SBoxes[1][0]);
    std::cout << SBoxes << "\n";
    std::cout << &SBoxes << "\n";
    std::cout << &(SBoxes[1]) << "\n";
    std::cout << ptrTest << "\n";
    std::cout << &(SBoxes[1][0]) << "\n";
    std::cout << (int)*ptrTest << "\n";
    std::cout << (int)*(ptrTest + 1) << "\n";
    std::cout << (int)SBoxes[1][0] << "\n";
    std::cout << (int)SBoxes[1][1] << "\n";

    // copy test
    unsigned char* test = (unsigned char*)malloc(64 * 8);
    //memcpy(test, SBoxes, 64*8);

    // copying each row of 64
    unsigned char* temp1, * temp2;
    for (int i = 0; i < 8; i++)
    {
        temp1 = test + i * 64;
        temp2 = &SBoxes[i][0];
        memcpy(temp1, temp2, 64);
    }

    // printout
    for (int i = 0; i < 8; i++)
    {
        std::cout << (int)test[i * 64] << ",";
    }
    std::cout << "\n";
}