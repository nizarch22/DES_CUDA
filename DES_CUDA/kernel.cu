#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "DES.h"
#include <stdio.h>
#include <iostream>

__global__ void EncryptDESCuda(uint64_t* messages, uint64_t* keys, unsigned char* matrices, unsigned char* sboxes, uint64_t* results);
__global__ void DecryptDESCuda(uint64_t* encryptions, uint64_t* keys, unsigned char* matrices, unsigned char* sboxes, uint64_t* results);
__global__ void EncryptDESCudaDebug(uint64_t* messages, uint64_t* keys, unsigned char* matrices, unsigned char* sboxes, uint64_t* results, uint64_t* debug, int n);
void EncryptDESDebug(const uint64_t& plaintext, const uint64_t& key, uint64_t& encryption, uint64_t* debug);
void printCharMatrix(unsigned char* matrix, int y, int x);

// Checks cuda errors. Exits if detected. 
// This may be helpful in release mode, where the kernel may not run if we demand too many resources.
#define CHECK_CUDA_ERROR(call) \
{ \
    hipError_t err = call; \
    if (err != hipSuccess) \
    { \
        fprintf(stderr, "CUDA Error: %s (error code %d) at %s:%d\n", \
                hipGetErrorString(err), err, __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} 

int main()
{
    // kernel parameters
    const int numThreads = 256;
    const int numMessages = 524288;// 524288 -  4MB - 10x speedup. 33554432 - 256MB - 70x speedup!
    const int numBlocks = (numMessages + numThreads - 1) / numThreads;

    // size parameters
    int bytesMessages = sizeof(uint64_t) * numMessages;
    int bytesKeys = sizeof(uint64_t) * numMessages;

    //// Kernel arguments prep stage ////
    // prep matrices, sboxes
    unsigned char* d_SBoxes, * d_matrices;
    unsigned char* matrices[7] = {IP,PC1,PC2, E, PMatrix,IPInverse, LCS};
    int matricesSizes[7] = { 64,56,48,48,32,64,16 };
    // prep keys, messages, encryptions, decryptions
    uint64_t* d_messages, * d_keys;
    uint64_t* messages = (uint64_t*)malloc(bytesMessages);
    uint64_t* keys = (uint64_t*)malloc(bytesKeys);
    for (int i = 0; i < numMessages; i++)
    {
        messages[i] = (((uint64_t)rand()) << 32) | rand();
        keys[i] = (((uint64_t)rand()) << 32) | rand();
    }
    
    // prep results
    uint64_t* d_resultsEncryption, * d_resultsDecryption;
    uint64_t* resultsEncryption = (uint64_t*)malloc(bytesMessages);
    uint64_t* resultsDecryption = (uint64_t*)malloc(bytesMessages);
    // CPU-run DES Results
    uint64_t* encryptions = (uint64_t*)malloc(bytesMessages);
    uint64_t* decryptions = (uint64_t*)malloc(bytesMessages);

    int startTimeAlloc = clock(); // Used to measure the time GPU finishes execution since allocation started.
    // cuda allocate memory - matrices, sboxes
    const int matricesSize = 328;
    const int sboxesSize= 512;
    hipMalloc(&d_matrices, matricesSize);
    hipMalloc(&d_SBoxes, sboxesSize);
    // cuda allocate memory - messages, keys
    hipMalloc(&d_messages, bytesMessages);
    hipMalloc(&d_keys, bytesKeys);
    // cuda allocate memory - results
    hipMalloc(&d_resultsEncryption, bytesMessages);
    hipMalloc(&d_resultsDecryption, bytesMessages);

    int startTimeCopy = clock(); // Used to measure the time GPU finishes execution since copying started.
    // cuda copy memory - matrices, sboxes
    hipMemcpy(d_SBoxes, &SBoxes[0][0], 64*8, hipMemcpyHostToDevice);
    int offset = 0;
    for (int i = 0; i < 7; i++)
    {
        hipMemcpy(d_matrices + offset, &matrices[i][0], matricesSizes[i], hipMemcpyHostToDevice);
        offset += matricesSizes[i];
    }
    // cuda copy memory - messages, keys
    hipMemcpy(d_messages, messages, bytesMessages, hipMemcpyHostToDevice);
    hipMemcpy(d_keys, keys, bytesKeys, hipMemcpyHostToDevice);

    //// Run Encryption & Decryption in CUDA stage ////
    // We encrypt the messages using EncryptDESCuda. Then, we use all those encrypted messages to run DecryptDESCuda.
    EncryptDESCuda << <numBlocks, numThreads >> > (d_messages, d_keys, d_matrices, d_SBoxes, d_resultsEncryption);
    hipDeviceSynchronize(); // wait for encrypt to finish
    DecryptDESCuda << <numBlocks, numThreads >> > (d_resultsEncryption, d_keys, d_matrices, d_SBoxes, d_resultsDecryption);
    
    // cuda copy results 
    hipMemcpy(resultsEncryption, d_resultsEncryption, bytesMessages, hipMemcpyDeviceToHost);
    hipMemcpy(resultsDecryption, d_resultsDecryption, bytesMessages, hipMemcpyDeviceToHost);
    int endTimeGPU = clock();

    // cuda check for errors in CUDA execution
    CHECK_CUDA_ERROR(hipGetLastError());


    //// Runtime measurement and calculation stage ////
    // Calculate timings for CUDA, CPU execution. 
    // CUDA has 2 timing calculations: one with allocation time and one without. The reason is that the allocation time is very big, and impactful for small input data (where CPU performs better than the GPU).

    int startTimeCPU = clock();
    for (int i = 0; i < numMessages; i++)
    {
        EncryptDES(messages[i], keys[i], encryptions[i]);
        DecryptDES(encryptions[i], keys[i], decryptions[i]);
    }
    int endTimeCPU = clock();
    int CPUTime = endTimeCPU - startTimeCPU;
    int CUDATime = endTimeGPU - startTimeAlloc;
    int CUDATimeCopy = endTimeGPU - startTimeCopy;

    // printout of timing results
    std::cout << "CUDA Debug results:\n";
    std::cout << "Total messages size: " << (numMessages >> 17) << "MB\n";
    std::cout << "Total time to allocate memory + copy memory back and forth:\n";
    std::cout << "GPU: " << CUDATime << "ms\n";
    std::cout << "CPU: " << CPUTime << "ms\n";
    std::cout << "GPU - only since copying: " << CUDATimeCopy << "ms\n";
    double speedup = (float)CPUTime / CUDATime;
    double speedupCopy = (float)CPUTime / CUDATimeCopy;
    std::cout << "Total speedup: " << speedup << "\n";
    std::cout << "speedup without counting allocation: " << speedupCopy << "\n";

    
    //// GPU-CPU encryption-decryption validation stage ////
    bool bEqualDecrypt = 1; bool bEqualEncrypt = 1;
    for (int i = 0; i < numMessages; i++)
    {
        bEqualDecrypt &= (resultsDecryption[i] == messages[i]);
        if(!bEqualDecrypt)
        {
            std::cout << "Decryption-message comparison failed at " << i << "\n";
            std::cout << resultsDecryption[i] << " - ";
            std::cout << messages[i] << "\n";
            break;
        }

        bEqualEncrypt &= (resultsEncryption[i] == encryptions[i]);
        if (!bEqualEncrypt)
        {
            std::cout << "CPU-GPU Encryption comparison failed at " << i << "\n";
            std::cout << resultsDecryption[i] << " - ";
            std::cout << messages[i] << "\n";
            break;
        }

    }

    if (bEqualDecrypt && bEqualEncrypt)
    {
        std::cout << "Success!\n";
    }

    //// Memory release stage ////

    // CPU
    free(messages);
    free(keys);
    free(resultsEncryption);
    free(resultsDecryption);
    free(encryptions);
    free(decryptions);
    // GPU/CUDA
    CHECK_CUDA_ERROR(hipFree(d_matrices));
    CHECK_CUDA_ERROR(hipFree(d_SBoxes));
    CHECK_CUDA_ERROR(hipFree(d_messages));
    CHECK_CUDA_ERROR(hipFree(d_keys));
    CHECK_CUDA_ERROR(hipFree(d_resultsEncryption));
    CHECK_CUDA_ERROR(hipFree(d_resultsDecryption));

    return 0;
}